#include "hip/hip_runtime.h"
#include "util.cuh"

__global__
void set_zero_double(double* x, unsigned int xdim) {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < xdim) x[idx] = 0.0f;
}

__global__
void set_zero_float(float* x, unsigned int xdim) {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < xdim) x[idx] = 0.0f;
}

__global__
void set_one_double(double* x, unsigned int xdim) {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < xdim) {
        x[idx] = 1.0f;
    }
}

__global__
void set_one_float(float* x, unsigned int xdim) {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < xdim) {
        x[idx] = 1.0f;
    }
}

void set_zeros_double(double* x, unsigned int xdim) {
    unsigned int blocks = std::ceil(double(xdim)/256);
    set_zero_double<<<blocks,256>>>(x,xdim);
}

void set_zeros_float(float* x, unsigned int xdim) {
    unsigned int blocks = std::ceil(float(xdim)/256);
    set_zero_float<<<blocks,256>>>(x,xdim);
}

void set_ones_double(double* x, unsigned int xdim) {
    unsigned int blocks = std::ceil(double(xdim)/256);
    set_one_double<<<blocks,256>>>(x,xdim);
}

void set_ones_float(float* x, unsigned int xdim) {
    unsigned int blocks = std::ceil(float(xdim)/256);
    set_one_float<<<blocks,256>>>(x,xdim);
}
