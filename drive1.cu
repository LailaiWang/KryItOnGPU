#include "arnoldi.cuh"
#include "MatDotVec.cuh"
#include "stdio.h"


// instantiate the function for debugging purpose
void (*print_on_gpu) (double*, unsigned int) = &print_data_wrapper<double>;

int driver1() {
    
    unsigned int n = 6;
    unsigned int m = 5;
    double *b = (double *) malloc(n*sizeof(double));
    double *q = (double *) malloc(n*sizeof(double));
    double *Q = (double *) malloc(n*m*sizeof(double));
    double *h = (double *) malloc(m*(m+1)*sizeof(double));
    double *v = (double *) malloc(n*sizeof(double));

    double *b_d;
    double *q_d;
    double *Q_d;
    double *h_d;
    double *v_d;
    
    hipMalloc((void**) &b_d, n*sizeof(double));
    hipMalloc((void**) &q_d, n*sizeof(double));
    hipMalloc((void**) &Q_d, n*m*sizeof(double));
    hipMalloc((void**) &h_d, m*(m+1)*sizeof(double));
    hipMalloc((void**) &v_d, n*sizeof(double));

    for (int i=0;i<n;i++) {
        b[i] = 1.0;
    }

    hipMemcpy(b_d, b, n*sizeof(double), hipMemcpyHostToDevice);
    
    // initialize the function pointer
    void (*fn) (double*, double*, unsigned int) = &MatDotVec_wrapper<double>;
    arnoldi<double>(fn, b_d, q_d, Q_d, h_d, v_d, n, m);
    
    hipMemcpy(Q, Q_d, n*m*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h, h_d, m*(m+1)*sizeof(double), hipMemcpyDeviceToHost);

    free(b);
    free(q);
    free(Q);
    free(h);
    free(v);

    hipFree(b_d);
    hipFree(q_d);
    hipFree(Q_d);
    hipFree(h_d);
    hipFree(v_d);
    
    return 0;
}

