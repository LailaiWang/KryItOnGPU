#include "arnoldi.cuh"
#include "MatDotVec.cuh"
#include "gmres.cuh"
#include "stdio.h"
#include <vector>
#include "util.cuh"
#include <iostream>
#include "hip/hip_runtime.h"
#include "hipblas.h"

int driver1() {
    
    unsigned int xdim    = 6;
    unsigned int kspace  = 5;
    
    std::vector<double> bvec(xdim, 1.0);
    std::vector<double> Qvec(xdim*(kspace+1), 1.0);
    std::vector<double> hvec(kspace*(kspace+1), 1.0);
    std::vector<double> vvec(xdim, 1.0);

    double *b = bvec.data();
    double *Q = Qvec.data();
    double *h = hvec.data();
    double *v = vvec.data();

    double *b_d;
    double *Q_d;
    double *h_d;
    double *v_d;
    
    hipMalloc((void**) &b_d, xdim*sizeof(double));
    hipMalloc((void**) &Q_d, xdim*(kspace+1)*sizeof(double));
    hipMalloc((void**) &h_d, kspace*(kspace+1)*sizeof(double));
    hipMalloc((void**) &v_d, xdim*sizeof(double));

    for (int i=0;i<xdim;i++) {
        b[i] = 1.0;
    }

    hipMemcpy(b_d, b, xdim*sizeof(double), hipMemcpyHostToDevice);
    
    // initialize the function pointer
    void (*fn) (double*, double*, unsigned int) = &MatDotVec_wrapper<double>;
    arnoldi<double>(fn, b_d, Q_d, h_d, v_d, xdim, kspace);
    
    hipMemcpy(Q, Q_d, xdim*(kspace+1)*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h, h_d, kspace*(kspace+1)*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(b_d);
    hipFree(Q_d);
    hipFree(h_d);
    hipFree(v_d);
    
    return 0;
}

/*
int driver2() {

    unsigned int xdim    = 200;
    unsigned int kspace  = 200;
    
    std::vector<double> bvec(xdim, 1.0);
    std::vector<double> Qvec(xdim*(kspace+1), 0.0);
    std::vector<double> hvec(kspace*(kspace+1), 0.0);
    std::vector<double> vvec(xdim, 0.0);

    double *b = bvec.data();
    double *Q = Qvec.data();
    double *h = hvec.data();
    double *v = vvec.data();

    struct gmres_app_ctx<double> gctx(
        xdim, kspace, 1e-10, 1e-9, 
        &allocate_ram_gmres_app_ctx_d,
        &deallocate_ram_gmres_app_ctx_d
    );
    
    gctx.allocate_ram(
        gctx.b,  gctx.Q,  gctx.h,  gctx.v, 
        gctx.sn, gctx.cs, gctx.e1, gctx.beta,
        gctx.xdim, 
        gctx.kspace
    );

    hipMemcpy(gctx.b, b, xdim*sizeof(double), hipMemcpyHostToDevice);

    double *x_d, *res_d;
    hipMalloc((void**) &x_d,   xdim*sizeof(double));
    hipMalloc((void**) &res_d, xdim*sizeof(double));
    struct precon_app_ctx<double> pctx(x_d, res_d, xdim);
    
    struct cublas_app_ctx bctx (&initialize_cublas, &finalize_cublas);
    bctx.create_cublas(&bctx.handle);
    
    void (*fn) (double*, double*, unsigned int) = &MatDotVec_wrapper<double>;
    void (*fprecon) (void*, double*) = &MFPreconditioner<double>;

    void (*gmresSol) (
        void (*) (double*, double*, unsigned int),
        void (*) (void*, double*),
        void*,
        void*,
        void*
    ) = &MFgmres<double>;

    gmresSol(fn, fprecon, (void *) &ptxaddr, (void *) &ctxaddr, (void *) &btxaddr);
    
    hipMemcpy(Q, gctx.Q, xdim*(kspace+1)*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h, gctx.h, kspace*(kspace+1)*sizeof(double), hipMemcpyDeviceToHost);

    std::vector<double> soln(xdim, 0.0);

    hipMemcpy(soln.data(), x_d, xdim*sizeof(double), hipMemcpyDeviceToHost);
    
    for(auto &d : soln) std::cout<<"sol "<<d<<std::endl;

    bctx.clean_cublas(&bctx.handle);

    gctx.deallocate_ram(
        gctx.b, gctx.Q, gctx.h, gctx.v, 
        gctx.sn, gctx.cs, gctx.e1, gctx.beta
    );
    
    hipFree(x_d);
    hipFree(res_d);
    return 0;
}
*/
