#include "gmres_ctx.cuh"
#include "cuda_constant.cuh"
#include "util.cuh"
#include <iostream>

void allocate_ram_gmres_app_ctx(
        double* &b,  double* &Q,  double* &h,  double* &v,
        double* &sn, double* &cs, double* &e1, double* &beta,
        unsigned int xdim, unsigned int kspace
){
    
    hipMalloc((void **) &b,    sizeof(double)*xdim);
    hipMalloc((void **) &Q,    sizeof(double)*xdim*(kspace+1));
    hipMalloc((void **) &h,    sizeof(double)*kspace*(kspace+1));
    hipMalloc((void **) &v,    sizeof(double)*xdim);

    hipMalloc((void **) &sn,   sizeof(double)*(kspace+1));
    hipMalloc((void **) &cs,   sizeof(double)*(kspace+1));
    hipMalloc((void **) &e1,   sizeof(double)*(kspace+1));
    hipMalloc((void **) &beta, sizeof(double)*(kspace+11));
}
    
void deallocate_ram_gmres_app_ctx(
        double* &b,  double* &Q,  double* &h,  double* &v,
        double* &sn, double* &cs, double* &e1, double* &beta
){
    hipFree(b);
    hipFree(Q);
    hipFree(h);
    hipFree(v);

    hipFree(sn);
    hipFree(cs);
    hipFree(e1);
    hipFree(beta);
}

void set_ones_const() {
    float  fno = -1.0;
    float  fpo =  1.0;
    double dno = -1.0;
    double dpo =  1.0;
    
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(P_ONE_F), &fpo, sizeof(float), 0, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(N_ONE_F), &fno, sizeof(float), 0, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(P_ONE_D), &dpo, sizeof(double), 0, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(N_ONE_D), &dno, sizeof(double), 0, hipMemcpyHostToDevice));
    return;
}
